#include "hip/hip_runtime.h"

#include "functions/convolution.h"
#include "functions/pooling.h"
#include "hip/hip_runtime.h"
#include <math.h>

//compile command
// nvcc -G main.cu functions/convolution.cu functions/pooling.cu functions/tasklib.cu functions/tasklib.cu -o main

#define STB_IMAGE_IMPLEMENTATION
#include "tools/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "tools/stb_image_write.h"


// usage: main.exe path_to_img.jpg -c path_to_convolution_output.jpg -p path_to_pooling_output.jpg
// -c flag to perform convolution and provide a path to the output file
// -p flag to perform max pooling and provide a path to the output file
// -a flag to perform average pooling and provide a path to the output file

int deviceIdx = 0;
hipDeviceProp_t deviceProp;
	

int main(int argc, char* argv[]){
    char* file;
    int convolution_selected = 0;
    char* convolution_output;
    int max_pooling_selected = 0;
    char* max_pooling_output;
    int min_pooling_selected = 0;
    char* min_pooling_output;
    int average_pooling_selected = 0;
    char* average_pooling_output;

    //cuda related tasks
	hipSetDevice(deviceIdx);
	hipGetDeviceProperties(&deviceProp, deviceIdx);
	printf("GPU is %s, index set is %d\n",deviceProp.name, deviceIdx);
    printf("Device has max %d threads per block\n",deviceProp.maxThreadsPerBlock);

    //argument parsing
    if(argc<2){
        //not enough arguments provided
    }
    file = argv[1];

    //load arguments
    for(int i=2;i<argc;i++){
        if(strcmp(argv[i],"-c")==0){
            convolution_selected = 1;
            convolution_output = argv[i+1];
            //printf("selected convolution and saving to %s\n",argv[i+1]);
        } else if(strcmp(argv[i],"-p")==0){
            max_pooling_selected = 1;
            max_pooling_output = argv[i+1];
        } else if(strcmp(argv[i],"-a")==0){
            average_pooling_selected = 1;
            average_pooling_output = argv[i+1];
        } else if(strcmp(argv[i],"-m")==0){
            min_pooling_selected = 1;
            min_pooling_output = argv[i+1];
        }
    }

    
    int width, height, channels;
    // Load the image into a buffer
    unsigned char* imageData = stbi_load(file, &width, &height, &channels, 0);

    //Gpu memory pointer used only if a function has to execute kernels on the original image, which is not every function
    uint8_t* imageData_gpu = NULL;
    hipMalloc(&imageData_gpu,width*height*channels);
    hipMemcpy(imageData_gpu, imageData, width * height * channels, hipMemcpyHostToDevice);

    
    //Schedule all operations
    void* convolution_outbuffer = NULL;
    void* maxpool_outbuffer = NULL;
    void* minpool_outbuffer = NULL;
    void* avgpool_outbuffer = NULL;

    void* convolution_imgout = NULL;
    void* maxpool_imgout = NULL;
    void* minpool_imgout = NULL;
    void* avgpool_imgout = NULL;

    hipStream_t stream_convolution;
    hipStream_t stream_maxpool;
    hipStream_t stream_minpool;
    hipStream_t stream_avgpool;
    

    //perform convolution
    if(convolution_selected){
        hipStreamCreate(&stream_convolution);
        hipMalloc(&convolution_outbuffer,width*height*channels);
        convolution_imgout = malloc(width*height*channels);

        //perform convolution on GPU
        convolution_2d<<<16,32,0,stream_convolution>>>((uint8_t*)convolution_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,width*height*channels);

        hipMemcpyAsync(convolution_imgout, convolution_outbuffer, width * height * channels, hipMemcpyDeviceToHost, stream_convolution);
    }
    //perform pooling
    if(max_pooling_selected){
        hipStreamCreate(&stream_maxpool);
        hipMalloc(&maxpool_outbuffer,(width/2)*(height/2)*channels); //half the original size
        maxpool_imgout = malloc((width/2)*(height/2)*channels);

        image_pooling_max<<<16,32,0,stream_maxpool>>>((uint8_t*)maxpool_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,(width/2)*(height/2)*channels);

        hipMemcpyAsync(maxpool_imgout, maxpool_outbuffer, (width/2)*(height/2)*channels, hipMemcpyDeviceToHost, stream_maxpool);
    }
    if(min_pooling_selected){
        hipStreamCreate(&stream_minpool);
        hipMalloc(&minpool_outbuffer,(width/2)*(height/2)*channels); //half the original size
        minpool_imgout = malloc((width/2)*(height/2)*channels);

        image_pooling_min<<<16,32,0,stream_minpool>>>((uint8_t*)minpool_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,(width/2)*(height/2)*channels);
        hipMemcpyAsync(minpool_imgout, minpool_outbuffer, (width/2)*(height/2)*channels, hipMemcpyDeviceToHost, stream_minpool);
    }
    if(average_pooling_selected){
        hipStreamCreate(&stream_avgpool);
        hipMalloc(&avgpool_outbuffer,(width/2)*(height/2)*channels); //half the original size
        avgpool_imgout = malloc((width/2)*(height/2)*channels);

        image_pooling_average<<<16,32,0,stream_avgpool>>>((uint8_t*)avgpool_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,(width/2)*(height/2)*channels);
        hipMemcpyAsync(avgpool_imgout, avgpool_outbuffer, (width/2)*(height/2)*channels, hipMemcpyDeviceToHost, stream_avgpool);
    }

    //write images
    if(convolution_selected){
        hipStreamSynchronize(stream_convolution);
        int success = stbi_write_jpg(convolution_output, width, height, 3, convolution_imgout, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }
        
        // Clean up
        free(convolution_imgout);
        hipStreamDestroy(stream_convolution);
        hipFree(convolution_outbuffer);
    }

    if(max_pooling_selected){
        //write image
        hipStreamSynchronize(stream_maxpool);
        int success = stbi_write_jpg(max_pooling_output, width/2, height/2, 3, maxpool_imgout, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }

        // Clean up
        free(maxpool_imgout);
        hipStreamDestroy(stream_maxpool);
        hipFree(maxpool_outbuffer);
    }

    if(min_pooling_selected){
        //write image
        hipStreamSynchronize(stream_minpool);
        int success = stbi_write_jpg(min_pooling_output, width/2, height/2, 3, minpool_imgout, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }

        // Clean up
        free(minpool_imgout);
        hipStreamDestroy(stream_minpool);
        hipFree(minpool_outbuffer);
    }

    if(average_pooling_selected){
        //write image
        hipStreamSynchronize(stream_avgpool);
        int success = stbi_write_jpg(average_pooling_output, width/2, height/2, 3, avgpool_imgout, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }
        // Clean up
        free(avgpool_imgout);
        hipStreamDestroy(stream_avgpool);
        hipFree(avgpool_outbuffer);

    }



    //cleanup
    stbi_image_free(imageData);
    if(imageData_gpu!=NULL)hipFree(imageData_gpu);

}