#include "hip/hip_runtime.h"

#include "functions/convolution.h"
#include "functions/pooling.h"
#include "hip/hip_runtime.h"
#include <math.h>

//compile command
// nvcc -G main.cu functions/convolution.cu functions/tasklib.cu functions/tasklib.cu -o main

#define STB_IMAGE_IMPLEMENTATION
#include "tools/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "tools/stb_image_write.h"


// usage: main.exe path_to_img.jpg -c path_to_convolution_output.jpg -p path_to_pooling_output.jpg
// -c flag to perform convolution and provide a path to the output file
// -p flag to perform max pooling and provide a path to the output file
// -a flag to perform average pooling and provide a path to the output file

int deviceIdx = 0;
hipDeviceProp_t deviceProp;
	

int main(int argc, char* argv[]){
    char* file;
    int convolution_selected = 0;
    char* convolution_output;
    int max_pooling_selected = 0;
    char* max_pooling_output;
    int average_pooling_selected = 0;
    char* average_pooling_output;

    //cuda related tasks
	hipSetDevice(deviceIdx);
	hipGetDeviceProperties(&deviceProp, deviceIdx);
	//printf("GPU is %s, index set is %d\n",deviceProp.name, deviceIdx);
    //printf("Device has max %d threads per block\n",deviceProp.maxThreadsPerBlock);

    //argument parsing
    if(argc<2){
        //not enough arguments provided
    }
    file = argv[1];

    //load arguments
    for(int i=2;i<argc;i++){
        if(strcmp(argv[i],"-c")==0){
            convolution_selected = 1;
            convolution_output = argv[i+1];
            //printf("selected convolution and saving to %s\n",argv[i+1]);
        } else if(strcmp(argv[i],"-p")==0){
            max_pooling_selected = 1;
            max_pooling_output = argv[i+1];
        } else if(strcmp(argv[i],"-a")==0){
            average_pooling_selected = 1;
            average_pooling_output = argv[i+1];
        }
    }

    
    int width, height, channels;
    // Load the image into a buffer
    unsigned char* imageData = stbi_load(file, &width, &height, &channels, 0);

    //Gpu memory pointer used only if a function has to execute kernels on the original image, which is not every function
    uint8_t* imageData_gpu = NULL;
    hipMalloc(&imageData_gpu,width*height*channels);
    hipMemcpy(imageData_gpu, imageData, width * height * channels, hipMemcpyHostToDevice);

    
    //Schedule all operations
    void* convolution_outbuffer = NULL;
    void* maxpool_outbuffer = NULL;
    void* avgpool_outbuffer = NULL;



    //perform convolution
    if(convolution_selected){
        hipMalloc(&convolution_outbuffer,width*height*channels);

        //perform convolution on GPU
        convolution_2d<<<512,256>>>((uint8_t*)convolution_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,width*height*channels);
    }
    //perform pooling
    if(max_pooling_selected){
        hipMalloc(&maxpool_outbuffer,(width/2)*(height/2)*channels); //half the original size

        image_pooling_max<<<256,128>>>((uint8_t*)maxpool_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,(width/2)*(height/2)*channels);
    }
    if(average_pooling_selected){
        hipMalloc(&avgpool_outbuffer,(width/2)*(height/2)*channels); //Half the original size

        image_pooling_average<<<256,128>>>((uint8_t*)avgpool_outbuffer,(uint8_t*)imageData_gpu,width,height,channels,(width/2)*(height/2)*channels);
    }

    //synq devices
    hipDeviceSynchronize();

    //write images
    if(convolution_selected){
        //write image
        void* imageDataResult = malloc(width*height*channels);
        hipMemcpy(imageDataResult,convolution_outbuffer,width*height*channels,hipMemcpyDeviceToHost);

        int success = stbi_write_jpg(convolution_output, width, height, 3, imageDataResult, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }

        // Clean up
        free(imageDataResult);
        hipFree(convolution_outbuffer);
    }

    if(max_pooling_selected){
        //write image
        void* imageDataResult = malloc((width/2)*(height/2)*channels);
        hipMemcpy(imageDataResult,maxpool_outbuffer,(width/2)*(height/2)*channels,hipMemcpyDeviceToHost);

        int success = stbi_write_jpg(max_pooling_output, width/2, height/2, 3, imageDataResult, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }

        // Clean up
        free(imageDataResult);
        hipFree(maxpool_outbuffer);
    }

    if(average_pooling_selected){
        //write image
        void* imageDataResult = malloc((width/2)*(height/2)*channels);
        hipMemcpy(imageDataResult,avgpool_outbuffer,(width/2)*(height/2)*channels,hipMemcpyDeviceToHost);

        int success = stbi_write_jpg(average_pooling_output, width/2, height/2, 3, imageDataResult, 90); // 90 is the quality

        if (success) {
            //printf("Image saved successfully.\n");
        } else {
            printf("Failed to save image.\n");
        }
        // Clean up
        free(imageDataResult);
        hipFree(avgpool_outbuffer);

    }



    //cleanup
    stbi_image_free(imageData);
    if(imageData_gpu!=NULL)hipFree(imageData_gpu);

}