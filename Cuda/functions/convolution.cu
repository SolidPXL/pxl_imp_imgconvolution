#include "hip/hip_runtime.h"
#include "convolution.h"
#include "tasklib.h"
#include <stdio.h>

__device__ float convolution_matrix[3][3]={{-1.0f,1.0f,-1.0f},{1.0f,0.0f,1.0f},{-1.0f,1.0f,-1.0f}};

void black_borders(uint8_t* dest, uint8_t* src, int width, int height, int channels){
    //set buffer to black
    int new_width = width+4;
    int new_height = height + 4;

    memset(dest,0,new_height*new_width);

    //copy row per row
    for(int i=0;i<height;i++){
        uint8_t* destStartAddr = dest + ((i+2)*new_width*channels) + 6;
        uint8_t* sourceStartAddr = src + (i*width*channels);
        memcpy(destStartAddr,sourceStartAddr,width*channels);
    }
}


__global__ void convolution_2d(uint8_t* dest, uint8_t* src, int width, int height,int channels, int N){

    for (int i = (blockIdx.x * blockDim.x + threadIdx.x);i < N;i += blockDim.x * gridDim.x) 
    {
        int row = i/(width*channels);
        int column = (i-(row*width*channels))/channels;
        int channel = i%channels;

        dest[i] = 
        convolution_matrix[0][0]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column-1,0,width),width,channels)+channel] +
        convolution_matrix[0][1]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column  ,0,width),width,channels)+channel] +
        convolution_matrix[0][2]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column+1,0,width),width,channels)+channel] +
        convolution_matrix[1][0]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column-1,0,width),width,channels)+channel] +
        convolution_matrix[1][1]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column  ,0,width),width,channels)+channel] +
        convolution_matrix[1][2]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column+1,0,width),width,channels)+channel] +
        convolution_matrix[2][0]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column-1,0,width),width,channels)+channel] +
        convolution_matrix[2][1]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column  ,0,width),width,channels)+channel] +
        convolution_matrix[2][2]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column+1,0,width),width,channels)+channel];
    }
    
}