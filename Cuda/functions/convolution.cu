#include "hip/hip_runtime.h"
#include "convolution.h"
#include "tasklib.h"
#include <stdio.h>

__device__ float convolution_matrix[3][3]={{-1.0f,1.0f,-1.0f},{1.0f,0.0f,1.0f},{-1.0f,1.0f,-1.0f}};

void black_borders(uint8_t* dest, uint8_t* src, int width, int height, int channels){
    //set buffer to black
    int new_width = width+4;
    int new_height = height + 4;

    memset(dest,0,new_height*new_width);

    //copy row per row
    for(int i=0;i<height;i++){
        uint8_t* destStartAddr = dest + ((i+2)*new_width*channels) + 6;
        uint8_t* sourceStartAddr = src + (i*width*channels);
        memcpy(destStartAddr,sourceStartAddr,width*channels);
    }
}


__global__ void convolution_2d(RGB* dest, RGB* src, int width, int height, int N){
    // int idx = (stridesize * stride) + (blockIdx.x * blockDim.x + threadIdx.x);
    // int i = idx*byte_per_pixel;
    // if(i>N) return;

    for (int i = (blockIdx.x * blockDim.x + threadIdx.x);i < N;i += blockDim.x * gridDim.x) 
    {
        int row = i/width;
        int column = i-(row*width);

        dest[i].r = 
        convolution_matrix[0][0]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column-1,0,width),width)].r +
        convolution_matrix[0][1]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column  ,0,width),width)].r +
        convolution_matrix[0][2]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column+1,0,width),width)].r +
        convolution_matrix[1][0]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column-1,0,width),width)].r +
        convolution_matrix[1][1]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column  ,0,width),width)].r +
        convolution_matrix[1][2]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column+1,0,width),width)].r +
        convolution_matrix[2][0]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column-1,0,width),width)].r +
        convolution_matrix[2][1]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column  ,0,width),width)].r +
        convolution_matrix[2][2]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column+1,0,width),width)].r;

        dest[i].g = 
        convolution_matrix[0][0]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column-1,0,width),width)].g +
        convolution_matrix[0][1]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column  ,0,width),width)].g +
        convolution_matrix[0][2]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column+1,0,width),width)].g +
        convolution_matrix[1][0]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column-1,0,width),width)].g +
        convolution_matrix[1][1]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column  ,0,width),width)].g +
        convolution_matrix[1][2]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column+1,0,width),width)].g +
        convolution_matrix[2][0]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column-1,0,width),width)].g +
        convolution_matrix[2][1]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column  ,0,width),width)].g +
        convolution_matrix[2][2]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column+1,0,width),width)].g;

        dest[i].b = 
        convolution_matrix[0][0]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column-1,0,width),width)].b +
        convolution_matrix[0][1]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column  ,0,width),width)].b +
        convolution_matrix[0][2]*src[getPixel_gpu(clamp_gpu(row-1,0,height),clamp_gpu(column+1,0,width),width)].b +
        convolution_matrix[1][0]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column-1,0,width),width)].b +
        convolution_matrix[1][1]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column  ,0,width),width)].b +
        convolution_matrix[1][2]*src[getPixel_gpu(clamp_gpu(row  ,0,height),clamp_gpu(column+1,0,width),width)].b +
        convolution_matrix[2][0]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column-1,0,width),width)].b +
        convolution_matrix[2][1]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column  ,0,width),width)].b +
        convolution_matrix[2][2]*src[getPixel_gpu(clamp_gpu(row+1,0,height),clamp_gpu(column+1,0,width),width)].b;
    }
    
}