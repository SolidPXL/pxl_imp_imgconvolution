#include "hip/hip_runtime.h"
#include "convolution.h"
#include "tasklib.h"

__device__ float convolution_matrix[3][3]={{-1.0f,1.0f,-1.0f},{1.0f,0.0f,1.0f},{-1.0f,1.0f,-1.0f}};

void black_borders(uint8_t* dest, uint8_t* src, int width, int height, int channels){
    //set buffer to black
    int new_width = width+4;
    int new_height = height + 4;

    memset(dest,0,new_height*new_width);

    //copy row per row
    for(int i=0;i<height;i++){
        uint8_t* destStartAddr = dest + ((i+2)*new_width*channels) + 6;
        uint8_t* sourceStartAddr = src + (i*width*channels);
        memcpy(destStartAddr,sourceStartAddr,width*channels);
    }
}


__global__ void convolution_2d(uint8_t* imgData, int width, int height, int channels){
    uint8_t* result;
    hipMalloc(&result,width*height*channels);
    for(int i=1;i<height-1;i++){
        //rows
        for(int j=1;j<width-1;j++){
            //columns
            for(int k=0;k<channels;k++){
                //channels

                result[getPixel_gpu(i,j,width,channels)+k] = 
                (imgData[getPixel_gpu(i-1,j-1,width,channels)+k]*convolution_matrix[0][0]) +
                (imgData[getPixel_gpu(i-1,j  ,width,channels)+k]*convolution_matrix[0][1]) +
                (imgData[getPixel_gpu(i-1,j+1,width,channels)+k]*convolution_matrix[0][2]) +
                (imgData[getPixel_gpu(i  ,j-1,width,channels)+k]*convolution_matrix[1][0]) +
                (imgData[getPixel_gpu(i  ,j  ,width,channels)+k]*convolution_matrix[1][1]) +
                (imgData[getPixel_gpu(i  ,j+1,width,channels)+k]*convolution_matrix[1][2]) +
                (imgData[getPixel_gpu(i+1,j-1,width,channels)+k]*convolution_matrix[2][0]) +
                (imgData[getPixel_gpu(i+1,j  ,width,channels)+k]*convolution_matrix[2][1]) +
                (imgData[getPixel_gpu(i+1,j+1,width,channels)+k]*convolution_matrix[2][2]);

            }
            

        }
    }
    memcpy(imgData,result,width*height*channels);
    free(result);
    
}