#include "hip/hip_runtime.h"
#include "pooling.h"
#include "tasklib.h"

#include <stdio.h>

__device__ int getMax(char* arr, int size){
    int max = *arr;
    for(int i=1;i<size;i++){
        if(arr[i]>max){
            max = arr[i];
        }
    }
    return max;
}

__device__ int getAverage(char* arr, int size){
    int average = *arr;
    for(int i=1;i<size;i++){
        average += (arr[i] - average) / i;
    }
    return average;
}



__global__ void image_pooling_max(uint8_t* dest, uint8_t* imgData, int width, int height, int channels){
    for (int i=0;i<height/2;i++){
        for(int j=0;j<width/2;j++){
            //printf("on i %d j %d\n",i,j);
            //Red channel
            char pool[4];
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)];

            dest[getPixel_gpu(i,j,width/2,channels)] = getMax(pool,4);

            //Green channel
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)+1];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)+1];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)+1];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)+1];

            dest[getPixel_gpu(i,j,width/2,channels)+1] = getMax(pool,4);

            //Blue channel
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)+2];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)+2];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)+2];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)+2];

            dest[getPixel_gpu(i,j,width/2,channels)+2] = getMax(pool,4);

        }
    }
    
}

__global__ void image_pooling_average(uint8_t* dest, uint8_t* imgData, int width, int height, int channels){
    for (int i=0;i<height/2;i++){
        for(int j=0;j<width/2;j++){
            //printf("on i %d j %d\n",i,j);
            //Red channel
            char pool[4];
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)];

            dest[getPixel_gpu(i,j,width/2,channels)] = getAverage(pool,4);

            //Green channel
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)+1];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)+1];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)+1];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)+1];

            dest[getPixel_gpu(i,j,width/2,channels)+1] = getAverage(pool,4);

            //Blue channel
            pool[0] = imgData[getPixel_gpu(i*2,j*2,width,channels)+2];
            pool[1] = imgData[getPixel_gpu((i*2)+1,j*2,width,channels)+2];
            pool[2] = imgData[getPixel_gpu(i*2,(j*2)+1,width,channels)+2];
            pool[3] = imgData[getPixel_gpu((i*2)+1,(j*2)+1,width,channels)+2];

            dest[getPixel_gpu(i,j,width/2,channels)+2] = getAverage(pool,4);

        }
    }
}