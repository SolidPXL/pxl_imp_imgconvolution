#include "hip/hip_runtime.h"
#include "pooling.h"
#include "tasklib.h"

#include <stdio.h>

__device__ int getMax(char* arr, int size){
    int max = *arr;
    for(int i=1;i<size;i++){
        if(arr[i]>max){
            max = arr[i];
        }
    }
    return max;
}

__device__ int getMin(char* arr, int size){
    int min = *arr;
    for(int i=1;i<size;i++){
        if(arr[i]<min){
            min = arr[i];
        }
    }
    return min;
}

__device__ int getAverage(char* arr, int size){
    int average = *arr;
    for(int i=1;i<size;i++){
        average += (arr[i] - average) / i;
    }
    return average;
}



__global__ void image_pooling_max(uint8_t* dest, uint8_t* src, int width, int height,int channels, int N){

    for (int i = (blockIdx.x * blockDim.x + threadIdx.x);i < N;i += blockDim.x * gridDim.x) 
    {
        char pool[4];
        int row = i/((width/2)*channels);
        int column = (i-(row*(width/2)*channels))/channels;
        int channel = i%channels;

        pool[0] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[1] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];
        pool[2] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[3] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];

        dest[getPixel_gpu(row,column,width/2,channels)+channel] = getMax(pool,4);
    }
    
}

__global__ void image_pooling_min(uint8_t* dest, uint8_t* src, int width, int height,int channels, int N){
    for (int i = (blockIdx.x * blockDim.x + threadIdx.x);i < N;i += blockDim.x * gridDim.x) 
    {
        char pool[4];
        int row = i/((width/2)*channels);
        int column = (i-(row*(width/2)*channels))/channels;
        int channel = i%channels;

        pool[0] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[1] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];
        pool[2] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[3] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];

        dest[getPixel_gpu(row,column,width/2,channels)+channel] = getMin(pool,4);
    }
    
}

__global__ void image_pooling_average(uint8_t* dest, uint8_t* src, int width, int height,int channels, int N){
    for (int i = (blockIdx.x * blockDim.x + threadIdx.x);i < N;i += blockDim.x * gridDim.x) 
    {
        char pool[4];
        int row = i/((width/2)*channels);
        int column = (i-(row*(width/2)*channels))/channels;
        int channel = i%channels;

        pool[0] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[1] = src[getPixel_gpu(clamp_gpu(2*row,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];
        pool[2] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu(2*column,0,width),width,channels)+channel];
        pool[3] = src[getPixel_gpu(clamp_gpu((2*row)+1,0,height),clamp_gpu((2*column)+1,0,width),width,channels)+channel];

        dest[getPixel_gpu(row,column,width/2,channels)+channel] = getAverage(pool,4);
    }
}